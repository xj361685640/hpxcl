
#include <hip/hip_runtime.h>
extern "C" __global__ void kernel(char *out, int *width, int *height, int *numIterations){
	unsigned int xDim = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yDim = blockIdx.y * blockDim.y + threadIdx.y;

	//index of the output array, multiplied by 3 for R,G,B values
	int arrayIndex = 3 * (*width) * yDim + xDim*3;

	float xPoint = ((float) xDim/(*width));
	float yPoint = ((float) yDim/(*width)); 

	//for calculation of complex number
	float x = 0;
	float y = 0;

	int iterationCount = 0;
	//terminating condition x^2+y^2 < 4 or iterations >numIterations
	while(y*y+x*x<=4 && iterationCount<(*numIterations)){
		float xTemp = x*x-y*y + xPoint;
		y = 2*x*y + yPoint;
		x = xTemp;
		iterationCount++;
	}

	if(iterationCount == (*numIterations)){
		out[arrayIndex] = 0;
		out[arrayIndex+1]=0;
		out[arrayIndex+2]=0;
	}else{
		out[arrayIndex] = iterationCount;
		out[arrayIndex+1]=iterationCount;
		out[arrayIndex+2]=iterationCount;
	}
}